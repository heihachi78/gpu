#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "prelude.h"
#include <chrono>

__global__ void mxm_naive_kernel(int N, float* a, float* b, float* c)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if(i>=N || j>=N)
    {
        return;
    }

    float tmp = 0;
    for (int k=0; k<N; k++){
        tmp += a[i*N+k] * b[k*N+j];
    }
    c[i * N + j] = tmp;
}

void mxm_serial(int N, float* a, float* b, float* c)
{
    // every item of the result
    for(int i=0; i<N; i++)
    {
        // is equal to the sum of every row
        for(int j=0; j<N; j++)
        {
            // multiplied by every column
            float tmp = 0.0f;
            for(int k=0; k<N; k++)
            {
                tmp += a[i*N+k] * b[k*N+j];
            }
            c[i*N+j] = tmp;
        }
    }
}

void const_init(int N, float* mat, float init)
{
    // init every element to init value
    for(int i=0; i<N*N; i++)
    {
        mat[i] = init;
    }
}

void diag_init(int N, float* mat, float init)
{
    // init every element to 0
    for(int i=0; i<N*N; i++)
    {
        mat[i] = 0.0;
    }
    
    // init the diagonal to init value
    for(int i=0; i<N; i++)
    {
        mat[i*N+i] = init;
    }
}

void mxm_test_gpu(int N)
{
    // heap allocate memory
    float* a = new float[N*N];
    float* b = new float[N*N];
    float* c = new float[N*N];

    // init matrices
    const_init(N, a, 2.5);
    const_init(N, c, 0.0);
    diag_init(N, b, 1.0);

    // declare null pointers for GPU memory
    float* d_a;
    float* d_b;
    float* d_c;

    // allocate memory on gpu
    CUDA_ERROR_CHECK (hipMalloc(reinterpret_cast<void**>(&d_a), N*N*sizeof(float)));
    CUDA_ERROR_CHECK (hipMalloc(reinterpret_cast<void**>(&d_b), N*N*sizeof(float)));
    CUDA_ERROR_CHECK (hipMalloc(reinterpret_cast<void**>(&d_c), N*N*sizeof(float)));

    // copy data to gpu
    CUDA_ERROR_CHECK (hipMemcpy(d_a, a, N*N*sizeof(float), hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK (hipMemcpy(d_b, b, N*N*sizeof(float), hipMemcpyHostToDevice));

    // initialize result variable on gpu
    CUDA_ERROR_CHECK (hipMemset(d_c, 0.0, N*N*sizeof(float)));

    // call the kernel
    dim3 block(32, 32);
    dim3 grid((N+block.x-1)/block.x, (N+block.y-1)/block.y);
    auto start = std::chrono::high_resolution_clock::now();
    mxm_naive_kernel<<<grid, block>>>(N, d_a, d_b, d_c);
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed_us = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

    CUDA_LASTERR();

    // copy data from device to memory
    CUDA_ERROR_CHECK (hipMemcpy(c, d_c, N*N*sizeof(float), hipMemcpyDeviceToHost));

    // printing the result
    printf("matrix size: %d, elapsed time on gpu: %f\n", N, float(elapsed_us)/1e3);

    // print the result matrix
    for(int i=0; i < N; i++)
    {
        for(int j=0; j < N; j++)
        {
            // printf("%0.2f ", c[i*N+j]);
        }
        // printf("\n");
    }

    // free memory
    CUDA_ERROR_CHECK (hipFree(d_a));
    CUDA_ERROR_CHECK (hipFree(d_b));
    CUDA_ERROR_CHECK (hipFree(d_c));
    delete[] a, b, c;
}

void mxm_test_serial(int N)
{
    // heap allocate memory
    float* a = new float[N*N];
    float* b = new float[N*N];
    float* c = new float[N*N];

    // init matrices
    const_init(N, a, 2.5);
    const_init(N, c, 0.0);
    diag_init(N, b, 1.0);

    // call our function and measure it
    auto start = std::chrono::high_resolution_clock::now();
    mxm_serial(N, a, b, c);
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed_us = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

    // printing the result
    printf("matrix size: %d, elapsed time on cpu: %f\n", N, float(elapsed_us)/1e3);

    // print the result matrix
    for(int i=0; i < N; i++)
    {
        for(int j=0; j < N; j++)
        {
            // printf("%0.2f ", c[i*N+j]);
        }
        // printf("\n");
    }

    // free memory
    delete[] a, b, c;
}

int main()
{
    for(int i=2; i<1024; i=i*2)
    {
        mxm_test_gpu(i);
        mxm_test_serial(i);
    }

    return 0;
}